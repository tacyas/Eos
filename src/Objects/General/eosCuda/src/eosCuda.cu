
#include <hip/hip_runtime.h>
/*
# %M% %Y% %I%
# The latest update : %G% at %U%
#
#%Z% eosCuda ver %I%
#%Z% Created by 
#%Z%
#%Z% Usage : eosCuda 
#%Z% Attention
#%Z%
*/

#ifdef CUDA

static char __sccs_id[] = "%Z%eosCuda ver%I%; Date:%D% %Z%";
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <stdlib.h>
#include <cuda_runtime.h>
#include "../inc/eosCuda.h"

#ifdef __cplusplus
} ;

#endif
int __eosCuda__=0;
int __eosCudaDeviceID__=0;
int __eosCudaBlockSize__=32;
int __eosCudaThreadsPerBlock__=1024;
static cudaDeviceProp __eosCudaDeviceProp__;

void
eosCudaInit(int id)
{
	cudaError_t error;

	__eosCuda__=1;
	__eosCudaDeviceID__ = id;

	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	cudaSetDevice(__eosCudaDeviceID__);
	error = cudaGetDevice(&__eosCudaDeviceID__);
	if (error != cudaSuccess) {
		fprintf(stderr, "cudaGetDevice returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = cudaGetDeviceProperties(&__eosCudaDeviceProp__, __eosCudaDeviceID__);
	if (__eosCudaDeviceProp__.computeMode == cudaComputeModeProhibited) {
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::cudaSetDevice().\n");
		exit(EXIT_SUCCESS);
	}
	if (error != cudaSuccess) {
		printf("cudaGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
	} 
    __eosCudaBlockSize__ = (__eosCudaDeviceProp__.major < 2) ? 16 : 32;
    __eosCudaThreadsPerBlock__ = __eosCudaDeviceProp__.maxThreadsPerBlock;
}


void
eosCudaPropertiesPrint(FILE* fpt, int id)
{
	cudaError_t error;

	__eosCuda__=1;
	__eosCudaDeviceID__ = id;

	// By default, we use device 0, otherwise we override the device ID based on what is provided at the command line
	cudaSetDevice(__eosCudaDeviceID__);
	error = cudaGetDevice(&__eosCudaDeviceID__);
	if (error != cudaSuccess) {
		fprintf(stderr, "cudaGetDevice returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = cudaGetDeviceProperties(&__eosCudaDeviceProp__, __eosCudaDeviceID__);
	if (__eosCudaDeviceProp__.computeMode == cudaComputeModeProhibited) {
		fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::cudaSetDevice().\n");
		exit(EXIT_SUCCESS);
	}
	if (error != cudaSuccess) {
		printf("cudaGetDeviceProperties returned error code %d, line(%d)\n", error, __LINE__);
		return;
	} 
    __eosCudaBlockSize__ = (__eosCudaDeviceProp__.major < 2) ? 16 : 32;

	fprintf(fpt, "GPU Device %d: \"%s\" with compute capability %d.%d\n\n", __eosCudaDeviceID__, __eosCudaDeviceProp__.name, __eosCudaDeviceProp__.major, __eosCudaDeviceProp__.minor);

	fprintf(fpt, "totalGlobalMem:     %d\n", __eosCudaDeviceProp__.totalGlobalMem);
	fprintf(fpt, "sharedMemPerBlock:  %d\n", __eosCudaDeviceProp__.sharedMemPerBlock);
	fprintf(fpt, "maxThreadsPerBlock: %d\n", __eosCudaDeviceProp__.maxThreadsPerBlock);
}

#endif
