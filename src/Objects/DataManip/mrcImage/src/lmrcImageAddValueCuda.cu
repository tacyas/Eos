#include "hip/hip_runtime.h"
/*
# lmrcImageAddValueCuda.cu : $Revision$  
# $Date$ 
# Created by $Author$
# Usage : lmrcImageAddValueCuda.cu 
# Attention
#   $Loccker$
#  	$State$ 
#
*/
/* $Log$ */
#include <stdio.h>
#include <hip/hip_runtime.h>
#undef DEBUG
#include "genUtil.h"
#include "eosCuda.h"
#include "./lmrcImageAddValue.h"

__global__ void lmrcImageAddValueKernel(float* im, float v, int max)
{
	int offset = threadIdx.x + blockIdx.x*blockDim.x;
	
	if(offset<max) {
		im[offset] = im[offset] + v;
	}
}

void 
lmrcImageAddValueCuda(float* out, float* in, float value, int nPixel, int imageMode, int mode)
{
	hipError_t err = hipSuccess;
	switch(imageMode) {
		case mrcFloatImage: {
			float* im;

			DEBUGPRINT3("%d %p -> %p\n", nPixel, in, out);
			err = hipMalloc((void **)&im,  sizeof(float)*nPixel);
			if(err !=  hipSuccess) {
				fprintf(stderr, "Memory Not Allocated\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMalloc\n");
			}
		
			hipMemcpy(im, in, sizeof(float)*nPixel, hipMemcpyHostToDevice);
			if(err !=  hipSuccess) {
				fprintf(stderr, "hipMemcpy\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMemcpy: HostToDevice\n");
			}

		    lmrcImageAddValueKernel<<<nPixel/__eosCudaThreadsPerBlock__,__eosCudaThreadsPerBlock__>>>(im, value, nPixel); 

			hipMemcpy(out, im, sizeof(float)*nPixel, hipMemcpyDeviceToHost);
			if(err !=  hipSuccess) {
				fprintf(stderr, "hipMemcpy\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMemcpy: DeviceToHost\n");
			}
			hipFree(im);

			break;
		}
		default: {
			fprintf(stderr, "Not supported Mode in lmrcImageAddValueCuda:  %d\n", imageMode); 
			exit(EXIT_FAILURE);
			break;
		}
	}	
}

