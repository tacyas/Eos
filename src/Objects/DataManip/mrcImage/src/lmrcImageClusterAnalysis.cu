#include "hip/hip_runtime.h"
/*
# %M% %Y% %I%
# The latest update : %G% at %U%
#
#%Z% mrcImageClusterAnalysis ver %I%
#%Z% Created by Narita  
#%Z% Modified by Tac for AutoRotationMode 
#%Z%
#%Z% Usage : mrcImageClusterAnalysis
#%Z% Attention
#%Z%
*/
static char __sccs_id[] = "%Z%mrcImageClusterAnalysis ver%I%; Date:%D% %Z%";
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>                  
#define DEBUG
#include "genUtil.h"
#include "Memory.h"
#include "./lmrcImageClusterAnalysis.h"


/*
	Historically, we created the mrcImageClusterAnalysis first.
	After this, we created the Cluster class in General.
	So, functions in this file have both of new and old system, especially prevM and prevN.
	In the future, we must modify the sources.
*/

#include "Cluster.h"

void
lmrcImageClusterAnalysisDistanceToLog(FILE* fpt, int n, int m, int prevM, int prevN, float distance)
{
	clusterLogOneRecord cluster;

	cluster.N = n;
	cluster.M = m;
	cluster.prevN = prevM; /* N-M exchange: Historical reason.*/
	cluster.prevM = prevN; /*  In the future, we must modify this inconsintency */
	cluster.distance = distance;
	clusterLogWriteOneRecord(&cluster, fpt, 0);
	fflush(fpt);
}

double
lmrcImageDistanceCalc0(mrcImage* out, mrcImage* cor, mrcImage* clusterN, mrcImage* clusterM, int numN, int numM, lmrcImageClusterAnalysisInfo* linfo) {
	double distance = 0;

	DEBUGPRINT("n lmrcImageDistanceCalc0:");
  	switch(linfo->mode) {
		case lmrcImageClusterAnalysisModeCorrelation:
		case lmrcImageClusterAnalysisModeLinearCorrelation:
		case lmrcImageClusterAnalysisModeEuclidDistance: {
		  if(linfo->flagAutoRotation) {
		  	distance = lmrcImageDistanceCalcWithAutoRotation(out, cor, 
							  clusterN, clusterM, 
			    			  &(linfo->arInitial), linfo->correlationMode, linfo->mode);
		  } else {
		  	DEBUGPRINT("ByCorrelation:");
	  	    distance = lmrcImageDistanceCalc(clusterN, clusterM, linfo->mode);
			*out = *clusterN;
			cor->Image = NULL;
		  }
		  break;
		}
		case lmrcImageClusterAnalysisModeWardMethod: 
		case lmrcImageClusterAnalysisModeWardMethodLinearCorrelation: {
		  if(linfo->flagAutoRotation) {
		  	distance = lmrcImageDistanceCalcByWardMethodWithAutoRotation(out, cor,
			                   clusterN,    clusterM, 
			                   numN, numM,
							   &(linfo->arInitial), linfo->correlationMode, linfo->mode);
		  } else {
		  	DEBUGPRINT("By Ward: ");
            distance = lmrcImageDistanceCalcByWardMethod(
							   clusterN, clusterM, 
		                       numN, numM, 
		                       linfo->mode);
			*out = *clusterN;
			cor->Image = NULL;
		  }
		  break;
		}
		default: {
		  fprintf(stderr, "Not supported mode: %d\n", linfo->mode);
		  exit(EXIT_FAILURE);
		  break;
		}
   	}
	DEBUGPRINT1("Distance: %f\n", distance)
	if(0<distance) {
		return distance;
	} else {
		return (0);
	}
} 

void 
lmrcImageClusterAnalysis(
	int* group,                         /* Output: Group Index */
	mrcImage* in,                       /* Input:  Images  */
	int N,                              /* Input:  Number of Images */ 
	int finalclusternum,                /* Input:  FinalClusterNum */
	lmrcImageClusterAnalysisInfo* linfo /* Input:  Information of ClusterAnalysis */)
{
  float** distance;             /* Distance Array */
  int* index;                   /* the least index of averaged images in each cluster */
  int* averagednum;             /* the number of averaged images in each cluster */
  int i, n,m,minn,minm,clusternum, indexMinN, indexMinM;
  int prevN;
  int prevM;
  float min,F;
  mrcImage* cluster;
  mrcImageParaTypeRealCoord offset;
  mrcImage out;
  mrcImage cor;
  mrcImage minOut;
  mrcImage minCor;
  lmrcImageAutoRotationCorrelationInfo minInfo; 
  int flagSkip   = 0;
  int flagNoValue= 1;
  int flagNoSkip = 0;
  clusterLog* clusterLog=NULL;

  minn       = minm = 0;
  clusternum = N; /* the current number of clusters */
  prevN      = N; /* previous N */
  prevM      = N; /* previous M */

/* Initial Stage */	
  /* Memory Allocation */   
  linfo->Distance = distance = matrix(0, N-1, 0, N-1);
  index       = ivector(0, N);
  averagednum = ivector(0, N-1);
  cluster              = (mrcImage *)memoryAllocate(sizeof(mrcImage)*N, "Enough memory is not available!");
  linfo->averagedImage = (mrcImage*)memoryAllocate(sizeof(mrcImage)*N, "Enough memory is not available!");

  if(linfo->flagLogIn) {
  	DEBUGPRINT("LogIn\n");
  	fseek(linfo->fptLogIn, 0L, SEEK_SET);
   	clusterLog = clusterLogReadAll(clusterLog, linfo->fptLogIn, 0); 
	clusterLogClusterNumberSet2(clusterLog, linfo->previousFileNumber, 0); 
	clusterLog->current=clusterLog->top;
	if(clusterLog->top->prevN==clusterLog->top->prevM) {
		DEBUGPRINT2("Previous FileNumber : %d ClusterNumber %d \n", clusterLog->top->prevN, clusterLog->top->clusterNumber);
		linfo->previousFileNumber = clusterLog->top->prevN;
	} else {
		fprintf(stderr, "Something wrong");
		exit(EXIT_FAILURE);
	}
  }

  offset.x = offset.y = offset.z =0;
  index[N] = N;
  for (i=0; i<N; i++) {
    index[i]       = i;
    averagednum[i] = 1;
    group[i]       = i;

    cluster[i].Header = in[0].Header;
    mrcInit(&cluster[i],0);
    lmrcImageCopy(&cluster[i], &in[i], offset);

    linfo->averagedImage[i].Header = in[0].Header;
    mrcInit(&linfo->averagedImage[i],0);
    lmrcImageCopy(&linfo->averagedImage[i], &cluster[i], offset);
  }
  minOut.Header = cluster[0].Header;
  minCor.Header = cluster[0].Header;
  mrcInit(&minOut, NULL);
  mrcInit(&minCor, NULL);

  /* Cluster Analysis */
  /* 
  	Calculation of distance[n][m] 
	  0 <=     n < clusternum
      0 <= m < n
              n\m 0 1 2 3 ...-2 cluseternum-1
              0   - - - - ... - -
	          1   * - - - ... - - 
	          2   * * - - ... - -
	          3   * * * - ... - -
	          .   * * * * ... - -
	          .   * * * * ... - -
	          .   * * * * ... - - 
   clusternum-1   * * * * ... * -  
         
   *: Calc
   -: No Calc
  */

  while (finalclusternum<clusternum) {
    min=MAXFLOAT;
    for (n=0; n<clusternum; n++) {
      for (m=0; m<n; m++) {

	  	/* Check whether correlation between index[n] and index[m] must be calculated. */	
  		if( clusternum != N       /* At the first step (N==clusternum), we cannot skip */
		 && index[n]   != index[prevM]          
	 	 && index[m]   != index[prevM]
		 && min <= distance[n][m]) {      
		 	flagSkip = 1;
		} else {
			flagSkip = 0;
		}
		if(linfo->flagLogIn && !flagSkip && !flagNoSkip) {
			clusterTypeReal d = -1;
			clusterLog->current = clusterLogGetDistance(clusterLog, n, m, clusternum - (N - linfo->previousFileNumber), &d);
			if(d<=0) {
				DEBUGPRINT3("Zero:: n: %d m: %d d: %g\n", n, m, d);
				flagNoSkip = 1; /* The log file has an error log line */
			}
			if(0<d) {
				distance[n][m] = d;
				flagSkip = 1;
			} else {
				flagSkip = 0;
			}
		} 

		/* Calc distance[n][m] */
		if(flagSkip && 0 < distance[n][m]) { 
			DEBUGPRINT4("skip:   n: %d m: %d prevM: %d prevN: %d\n", n, m, prevM, prevN);
			DEBUGPRINT4("index:  n: %d m: %d prevM: %d prevN: %d\n", index[n], index[m], index[prevM], index[prevN]);
			/* Skip because of no need of calculation */
		} else {
			DEBUGPRINT4("noskip: n: %d m: %d prevM: %d prevN: %d\n", n, m, prevM, prevN);
			DEBUGPRINT4("index:  n: %d m: %d prevM: %d prevN: %d\n", index[n], index[m], index[prevM], index[prevN]);
			distance[n][m] = lmrcImageDistanceCalc0(&out, &cor, 
				&cluster[index[n]], &cluster[index[m]], 
				averagednum[index[n]], averagednum[index[m]], linfo); 
		} 

		/* Evaluate Distance betweeen cluster[index[n]] and cluster[index[m]] */
		if(distance[n][m]<=min) { 
			min  = distance[n][m];
		    minn = n;
		    minm = m;
			if(flagSkip) {
				flagNoValue = 1;
			} else {
				flagNoValue = 0;
				mrcImageFree(&minCor, "in lmrcImageClusterAnalysis");
				mrcImageFree(&minOut, "in lmrcImageClusterAnalysis");
			  	minCor = cor;
			 	minOut = out;
				minInfo = linfo->arInitial;
			}
		} else {
			if(!flagSkip) {
				mrcImageFree(&cor, "in lmrcImageClusterAnalysis");
				mrcImageFree(&out, "in lmrcImageClusterAnalysis");
			}
		}
		lmrcImageClusterAnalysisDistanceToLog(linfo->fptLog, n, m, prevN, prevM, distance[n][m]); 
      }
    } /* End: Calc Distance */

#ifdef DEBUG
	/* Check For Debuging */
	for(n=0; n<clusternum; n++) {
	for(m=0; m<n; m++) {
		DEBUGPRINT1("%6.2f ", distance[n][m]);
	}
		DEBUGPRINT("\n");
	}
    DEBUGPRINT3("index[minm] %d index[minn] %d min %g\n",index[minm], index[minn], min);
    DEBUGPRINT2("minm        %d minn        %d \n",minm, minn);
#endif
    /* 
		cluster[index[minm]] : change
		cluster[index[minn]] : no change
		index[0:clusternum-1]
		index[minm]
		index[minn]
		averagednum
    */
	if(flagNoValue) {
		distance[minn][minm] = lmrcImageDistanceCalc0(&out, &cor,
			&cluster[index[minn]], &cluster[index[minm]], 
			averagednum[index[minn]], averagednum[index[minm]], linfo); 
		mrcImageFree(&minCor, "in lmrcImageClusterAnalysis");
		mrcImageFree(&minOut, "in lmrcImageClusterAnalysis");
		minCor = cor;
		minOut = out;
		minInfo = linfo->arInitial;
	}
	indexMinN = index[minn];
	indexMinM = index[minm];
    lmrcImageClusterMerge(&cluster[index[minm]], &minOut, index, averagednum, minm, minn, &clusternum);
	lmrcImageCopy(&linfo->averagedImage[indexMinN], &cluster[indexMinM], offset);
	lmrcImageCopy(&linfo->averagedImage[indexMinM], &cluster[indexMinM], offset);
	if(linfo->flagLog2) {
		LOGPRINT4(linfo->fptLog2, "CLUST", "MIN", "%5d %5d %15.6g %15.6g", indexMinN, indexMinM, min, minInfo.LinearCorrelation);
	}
	DEBUGPRINT("after\n");
    DEBUGPRINT3("index[minm] %d index[minn] %d min %g\n",index[minm], index[minn], min);

    /*
    distance[n][m]
              n\m 0 1 2 3 ...-1 cluseternum
              0   - - - - ... - -
	          1   * - - - ... - - 
	          2   * * - - ... - -
	          3   * * * - ... - -
	          .   * * * * ... - -
	       minm   * * * * ... - -
	          .   * * * * ... - - 
	       minn   * * * * ... - - 
	          .   * * * * ... - - 
     clusternum   * * * * ... * -  

    group[n]: the number of the group including the nth-image  
      ~                                         ~
	0<= n < N
    */

	/* Grouping : group[minm] <- group[minn] */
    for (n=0; n<N; n++){
      if (group[n] == minn) {
	    group[n] = minm;         /* assignment of the new number of group */
      } else if (group[n] > minn){
	    group[n] = group[n] - 1; /* decrement of the number of group      */
      }
    }

	prevM = minm; /* Always, minm < minn.  So prevM<prevN */ 
	prevN = minn;
    for (n=0; n<clusternum; n++) {
     	for (m=0; m<n; m++) {
			if(m<prevN) {
				if(n<prevN) {
					distance[n][m] = distance[n  ][m  ];
				} else if(prevN<=n){
					distance[n][m] = distance[n+1][m  ];
				}
			} else if(prevN<=m){
				if(n<prevN) {
					fprintf(stderr, "Something wrong: n<m.  Always, m<n\n");
				} else if(prevN<=n) {
					distance[n][m] = distance[n+1][m+1];
				}
			}
		}
	}
    DEBUGPRINT("After distance change\n");
    for(n=0; n<clusternum; n++) {
    for(m=0; m<n; m++) {
		DEBUGPRINT1("%6.2f ", distance[n][m]);
    }
		DEBUGPRINT("\n");
    }
  } /* while end : Cluster Analysis Main Routine */
	
  for (n=0; n<N; n++){
    if (linfo->flagIteration){
      fprintf(stderr, "n %d group %d \n", n, group[n]);
    } else {
      fprintf(stderr, "n %d group %d \n", n, group[n]);
    }
  }
  if (linfo->flagIteration){
  	F = lmrcImageClusterAnalysisByIteration(group, in, finalclusternum, N, linfo->mode);
    for (n=0; n<N; n++){
      fprintf(stderr, "%d %d \n",n,group[n]);
    }
    fprintf(stdout,"F= %g\n",F);
  }
}

/*
lmrcImageClusterMerge
	in1: index[m] -> an averaged image of a new cluster: 
	in2: index[n] -> no change
	index:
	averagednum: the number of averaged images in each cluster
	m: The nearest pair of particles 
	n:                        -> the index to be neglected 
	N: Current cluster number -> N - 1
*/
void lmrcImageClusterMerge(mrcImage* in1, mrcImage* in2, int* index, int* averagednum, int m, int n, int* N)
{
  int io,ii;
  mrcImage in1mul,in2mul;
  float avrm,avrn;
  mrcImageParaTypeRealCoord offset;

  /* Averaging of cluster index[m] and index[n] */
  avrm = (float)averagednum[index[m]] / (averagednum[index[m]] + averagednum[index[n]]);
  avrn = (float)averagednum[index[n]] / (averagednum[index[m]] + averagednum[index[n]]);
  lmrcImageMultiplying(&in1mul, in1, avrm, in1->HeaderMode);
  lmrcImageMultiplying(&in2mul, in2, avrn, in2->HeaderMode);
  llmrcImageAdd(&in1mul, &in2mul);

  /* Set a new averaged image of a new cluster */
  offset.x = offset.y = offset.z =0 ;
  lmrcImageCopy(in1, &in1mul, offset);

  /* Set the number of averaged images */
  averagednum[index[m]] += averagednum[index[n]];

  /* 
    cluster index change : 
		ii<n:  io == ii
		n <ii: io == ii - 1
  */ 
  io=0;
  for (ii=0; ii<*N; ii++) {
    if (ii != n) {
      index[io] = index[ii];
      io++;
    }
  }
  *N = *N - 1;

  /* cleaning */
  mrcImageFree(&in1mul,"");
  mrcImageFree(&in2mul,"");
}

float 
lmrcImageDistanceCalcWithAutoRotation(mrcImage* out, mrcImage* cor, mrcImage* in1, mrcImage* in2, lmrcImageAutoRotationCorrelationInfo* linfo, int lmode, int mode)
{
	double distance;
	switch(mode) {
		case lmrcImageClusterAnalysisModeCorrelation: {
			lmrcImageAutoRotationCorrelation(out, cor, in1, in2, linfo, lmode); 
			distance = -linfo->max;
			break;
		}
		case lmrcImageClusterAnalysisModeEuclidDistance: {
			lmrcImageAutoRotationCorrelation(out, cor, in1, in2, linfo, lmode); 
			distance = linfo->EuclidDistance;
			break;
		}
		case lmrcImageClusterAnalysisModeLinearCorrelation: {
			lmrcImageAutoRotationCorrelation(out, cor, in1, in2, linfo, lmode); 
			distance = linfo->LinearCorrelation;
			break;
		}
		default: {
			fprintf(stderr, "Not supported mode in lmrcImageDistanceCalcWithAutoRotation: %d\n", mode);
			exit(EXIT_FAILURE);
			break;
		}
	}
	return distance;
}

float 
lmrcImageDistanceCalc(mrcImage* in1, mrcImage* in2, int mode)
{
  double data;
  
  switch (mode) {
  	case lmrcImageClusterAnalysisModeCorrelation: {
    	data = -lmrcImageCorrelationinSpace(in1, in2, 0);
    	break;
	}
  	case lmrcImageClusterAnalysisModeEuclidDistance: {
		data = lmrcImageEuclidDistanceCalc(in1, in2, 1);
		break;
	}
  	case lmrcImageClusterAnalysisModeWardMethod: {
		data = lmrcImageEuclidDistanceCalc(in1, in2, 1);
		data = SQR(data);
		break;
	}
  	case lmrcImageClusterAnalysisModeWardMethodLinearCorrelation: 
	case lmrcImageClusterAnalysisModeLinearCorrelation: {
		data = lmrcImageLinearCorrelation(in1, in2, 1);
		data = exp(SQR(data-1.0)/(1/log(1e4)))-1.0;
		break;
	}	
	default: {
		fprintf(stderr, "Not supported mode in lmrcImageDistanceCalc: %d\n", mode);
		exit(EXIT_FAILURE);
		break;
	}
  }
  return data;
}

float 
lmrcImageDistanceCalcByWardMethodWithAutoRotation(mrcImage* out, mrcImage* cor, mrcImage* in1, mrcImage* in2, int num1, int num2, lmrcImageAutoRotationCorrelationInfo* linfo, int lmode, int mode)
{
	double distance;
	switch(mode) {
		case lmrcImageClusterAnalysisModeWardMethod: {
			lmrcImageAutoRotationCorrelation(out, cor, in1, in2, linfo, lmode); 
			distance = SQR(linfo->EuclidDistance)*(num1*num2)/(num1+num2);
			break;
		}
		case lmrcImageClusterAnalysisModeWardMethodLinearCorrelation: {
			lmrcImageAutoRotationCorrelation(out, cor, in1, in2, linfo, lmode);
			distance = (exp(SQR(linfo->LinearCorrelation-1.0)/(1/log(1e4)))-1.0)*(num1*num2)/(num1+num2);
			break;
		}
		default: {
			fprintf(stderr, "Not supported mode in lmrcImageDistanceCalcByWardMethodWithAutoRotation: %d\n", mode);
			exit(EXIT_FAILURE);
			break;
		}
	}
	return distance;
}

float 
lmrcImageDistanceCalcByWardMethod(mrcImage* in1, mrcImage* in2, int num1, int num2, int mode)
{
	double data;
	double distance;

	switch(mode) {
		case lmrcImageClusterAnalysisModeWardMethod: {
  			data = lmrcImageEuclidDistanceCalc(in1, in2, 0);
  			distance = (data)*((num1*num2)/(num1+num2));
			break;
		}
		case lmrcImageClusterAnalysisModeWardMethodLinearCorrelation: {
			data = lmrcImageLinearCorrelation(in1, in2, 0); 
			if(0<data) {
				distance = (200.0/(data+1.0)-100)*(num1*num2)/(num1+num2);
			} else {
				distance = (200.0/(data+1.0)-100)*(num1*num2)/(num1+num2);
			}
			break;
		}
		default: {
			fprintf(stderr, "Not supported mode in lmrcImageDistanceCalcByWardMethod: %d\n", mode);
			exit(EXIT_FAILURE);
			break;
		}
	}
  	return data;
}

float lmrcImageClusterAnalysisByIteration(int* group, mrcImage* in, int clusternum, int N, int mode)
{
  mrcImage* center;
  int n,m;
  float distancesum,distance,olddistancesum,distancemin,F;
  int* groupnum;
  mrcImageParaTypeRealCoord dum;
  mrcImage tmpimage;
  double clustervar;
  mrcImage centerofclusters;

  center = (mrcImage *)memoryAllocate(sizeof(mrcImage)*clusternum, "Enough memory is not available!");
  groupnum = ivector(0,clusternum-1);
  dum.x = dum.y = dum.z =0;

  distancesum=1;
  olddistancesum=0;
  for (n=0; n<clusternum; n++){
    center[n].Header = in[0].Header;
    mrcInit(&center[n],0);
  }
    
  while (distancesum != olddistancesum){
    olddistancesum=distancesum;
    distancesum=0;
    for (n=0; n<clusternum; n++){
      mrcImageFree(&center[n],"");
      center[n].Header = in[0].Header;
      mrcInit(&center[n],0);
      groupnum[n]=0;
    }
    for (n=0; n<N; n++){
      llmrcImageAdd(&center[group[n]],&in[n]);
      groupnum[group[n]] ++;
    }
    for (n=0; n<clusternum; n++){
      lmrcImageMultiplying(&tmpimage, &center[n], 1.0/groupnum[n], center[n].HeaderMode);
      lmrcImageCopy(&center[n], &tmpimage, dum);
      mrcImageFree(&tmpimage, "");
    }
    for (n=0; n<N; n++){
      distancemin = 1e+16;
      for (m=0; m<clusternum; m++){
	distance = lmrcImageDistanceCalc(&in[n], &center[m], mode);
	if (distance < distancemin){
	  distancemin = distance;	  
	  group[n] = m;
	}
      }
      distancesum += distancemin;
    }
  }

  /* F-value */
  centerofclusters.Header = center[0].Header;
  mrcInit(&centerofclusters,0);
  for (n=0; n<clusternum; n++){
    llmrcImageAdd(&centerofclusters, &center[n]);
  }
  lmrcImageMultiplying(&tmpimage, &centerofclusters, 1.0/clusternum, centerofclusters.HeaderMode);
  lmrcImageCopy(&centerofclusters, &tmpimage, dum);
  mrcImageFree(&tmpimage, "");
  clustervar=0;
  for (n=0; n<clusternum; n++){
    clustervar += lmrcImageDistanceCalc(&centerofclusters, &center[n], 2);
  }
  clustervar /= clusternum;
  if (mode != 2){
    distancesum = 0;
    for (n=0; n<N; n++){
      distancesum += lmrcImageDistanceCalc(&in[n], &center[group[n]], 2);
    }
  }
  distancesum/=N;
  if (distancesum > 0){
    F=clustervar/distancesum;
    F=sqrt(F);
  } else {
    F=0;
  }
  return(F);
}
