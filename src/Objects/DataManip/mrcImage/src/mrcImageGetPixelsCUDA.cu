#include "hip/hip_runtime.h"
/*
# %M% %Y% %I%
# The latest update : %G% at %U%
#
#%Z% mrcImageGetPixelsCUDA.cu ver %I%
#%Z% Created by 
#%Z%
#%Z% Usage : mrcImageGetPixelsCUDA.cu 
#%Z% Attention
#%Z%
*/
static char __sccs_id[] = "%Z%mrcImageGetPixelsCUDA.cu ver%I%; Date:%D% %Z%";

#include <stdio.h>
#include <hip/hip_runtime.h>
#define DEBUG
#include "genUtil.h"
#include "eosCuda.h"
#include "../inc/mrcImage.h"
#include "./mrcImageGetPixelsCUDA.h"


__device__ float mrcPixelDataGetDevice(float x, float y, float z, float* inD, int nx, int ny, int nz, mrcPixelDataHowToGet howToMode)
{
	float data;

	switch(howToMode) {
		case mrcPixelHowNearest: {
			int   ix, iy, iz;
			int   n;
			ix = (int)(x+0.5);
			iy = (int)(y+0.5);
			iz = (int)(z+0.5);

			n = ix + iy*nx + iz*nx*ny;
			data = inD[n];
			break;
		}
		case mrcPixelHowLinear: {
			int   ix0, iy0, iz0;
			int   ix1, iy1, iz1;

			if(1<ny && 1<nz) { // 3D  
				int n000, n001, n010, n011;	
				int n100, n101, n110, n111;	
				float data000, data010, data100, data110;
				float data001, data011, data101, data111;

				ix0 = (int)x;
				iy0 = (int)y;
				iz0 = (int)z;
				ix1 = ix0+1;
				iy1 = iy0+1;
				iz1 = iz0+1;

				n000 = ix0 + iy0*nx + iz0*nx*ny; 
				n010 = ix0 + iy1*nx + iz0*nx*ny;
				n100 = ix1 + iy0*nx + iz0*nx*ny;
				n110 = ix1 + iy1*nx + iz0*nx*ny;
				n001 = ix0 + iy0*nx + iz1*nx*ny; 
				n011 = ix0 + iy1*nx + iz1*nx*ny;
				n101 = ix1 + iy0*nx + iz1*nx*ny;
				n111 = ix1 + iy1*nx + iz1*nx*ny;

				data000 = inD[n000];
				data010 = inD[n010];
				data100 = inD[n100];
				data110 = inD[n110];
				data001 = inD[n001];
				data011 = inD[n011];
				data101 = inD[n101];
				data111 = inD[n111];

				data = (ix1 - x  )*(iy1 - y  )*(iz1 - z  )*data000 
					 + (ix1 - x  )*(y   - iy0)*(iz1 - z  )*data010
					 + (x   - ix0)*(iy1 - y  )*(iz1 - z  )*data100
				  	 + (x   - ix0)*(y   - iy0)*(iz1 - z  )*data110
					 + (ix1 - x  )*(iy1 - y  )*(z   - iz0)*data001 
					 + (ix1 - x  )*(y   - iy0)*(z   - iz0)*data011
					 + (x   - ix0)*(iy1 - y  )*(z   - iz0)*data101
				  	 + (x   - ix0)*(y   - iy0)*(z   - iz0)*data111;

			} else if(1<ny) { // 2D 
				int n00, n01, n10, n11;	
				float data00, data01, data10, data11;

				ix0 = (int)x;
				iy0 = (int)y;
				ix1 = ix0+1;
				iy1 = iy0+1;

				n00 = ix0 + iy0*nx;
				n01 = ix0 + iy1*nx;
				n10 = ix1 + iy0*nx;
				n11 = ix1 + iy1*nx;

				data00 = inD[n00];
				data01 = inD[n01];
				data10 = inD[n10];
				data11 = inD[n11];

				data = (ix1 - x  )*(iy1 - y  )*data00 
					 + (ix1 - x  )*(y   - iy0)*data01
					 + (x   - ix0)*(iy1 - y  )*data10
					 + (x   - ix0)*(y   - iy0)*data11;
			} else { // 1D
				int n0, n1;	
				float data0, data1;

				ix0 = (int)x;
				ix1 = ix0+1;

				n0 = ix0;
				n1 = ix1;
				data0 = inD[n0];
				data1 = inD[n1];

				data = (ix1 - x  )*data0 
				  	 + (x   - ix0)*data1; 
			}
			break;
		}
		case mrcPixelHowCubicConv: {
			break;
		}
		case mrcPixelHowPolySig: {
			break;
		}
		default: {
			break;
		}
	}
		
	return data;
}

__global__ void mrcPixelsDataGetKernel(float* outD, float* inCoordD, int num, float* inD, int nx, int ny, int nz, mrcPixelDataHowToGet howToMode)
{
	int offset = threadIdx.x + blockIdx.x*blockDim.x; 

	if(offset<num) {
		float x, y, z;

		x = inCoordD[3*offset+0];	
		y = inCoordD[3*offset+1];	
		z = inCoordD[3*offset+2];	

		outD[offset] = mrcPixelDataGetDevice(x, y, z, inD, nx, ny, nz, howToMode);
	}
}

__global__ void mrcPixelDataGetbyMatrix3DKernel(float* outD, float* matD, float* inD, int nx, int ny, int nz, mrcPixelDataHowToGet howToMode)
{
	int offset = threadIdx.x + blockIdx.x*blockDim.x; 
	int num = nx*ny*nz;

	if(offset<num) {
		float dstx, dsty, dstz;
		float srcx, srcy, srcz;
		float x, y, z;

		// offset = dstx + dsty*nx + dstz*nx*ny; // 3D
		// offset = dstx + dsty*nx;              // 2D
		// offset = dstx ;                       // 1D
		if(1<ny && 1<nz) { // 3D
			dstz = offset/(nx*ny);
			dsty = (offset - dstz*nx*ny)/nx;	
			dstx = (offset - dstz*nx*ny - dsty*nx);	
		} else if(1<ny) { // 2D
			dstz = 0;
			dsty = offset/nx;
			dstx = offset%nx;
		} else {          // 1D
			dstz = 0;
			dsty = 0;
			dstx = offset;
		}
		x = dstx - matD[3*4 + 0]; 
		y = dsty - matD[3*4 + 1];
		z = dstz - matD[3*4 + 2]; 
		srcx = matD[0*4+0]*x + matD[0*4+1]*y + matD[0*4+2]*z;
		srcy = matD[1*4+0]*x + matD[1*4+1]*y + matD[1*4+2]*z;
		srcz = matD[2*4+0]*x + matD[2*4+1]*y + matD[2*4+2]*z;
		outD[offset] = mrcPixelDataGetDevice(srcx, srcy, srcz, inD, nx, ny, nz, howToMode);
	}
}


void
mrcPixelsDataGetCUDA(float* out, float* inCoord, int num, float* in, mrcImageMode imageMode,  int nx, int ny, int nz, mrcPixelDataType type,  mrcPixelDataHowToGet howToMode)
{
	hipError_t err = hipSuccess;
	int nPixel = nx*ny*nz;
	
	DEBUGPRINT("mrcPixelsDataGetCUDA start ----> \n");	

	switch(imageMode) {
		case mrcFloatImage: {
			float* inD;
			float* outD;
			float* inCoordD;

			// inD
			DEBUGPRINT3("%d %p -> %p\n", nPixel, in, out);
			err = hipMalloc((void **)&inD,  sizeof(float)*nPixel);
			if(err !=  hipSuccess) {
				fprintf(stderr, "Memory Not Allocated\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMalloc\n");
			}
			// outD
			err = hipMalloc((void **)&outD,  sizeof(float)*num);
			if(err !=  hipSuccess) {
				fprintf(stderr, "Memory Not Allocated\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMalloc\n");
			}
			// inCoordD
			err = hipMalloc((void **)&inCoordD,  sizeof(float)*num*3);
			if(err !=  hipSuccess) {
				fprintf(stderr, "Memory Not Allocated\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMalloc\n");
			}
			
			// in -> inD
			hipMemcpy(inD, in, sizeof(float)*nPixel, hipMemcpyHostToDevice);
			// inCoord -> inD
			hipMemcpy(inCoordD, inCoord, sizeof(float)*3*num, hipMemcpyHostToDevice);

			if(err !=  hipSuccess) {
				fprintf(stderr, "hipMemcpy\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMemcpy: HostToDevice\n");
			}
			mrcPixelsDataGetKernel<<<num/__eosCudaThreadsPerBlock__,__eosCudaThreadsPerBlock__>>>(outD, inCoordD, num, inD, nx, ny, nz, howToMode);
			
			hipMemcpy(out, outD, sizeof(float)*num, hipMemcpyDeviceToHost);
			if(err !=  hipSuccess) {
				fprintf(stderr, "hipMemcpy\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMemcpy: DeviceToHost\n");
			}
			hipFree(inD);
			hipFree(inCoordD);
			hipFree(outD);
			break;
		}
		default: {
			fprintf(stderr, "Not supported mode in mrcPixelDataGetCUDA: %d\n", imageMode);
			exit(EXIT_FAILURE);
			break;
		}
	}
}


void
mrcPixelDataGetbyMatrix3DCUDA(float* out, float* mat, float* in, mrcImageMode imageMode, int nx, int ny, int nz, mrcPixelDataType type, mrcPixelDataHowToGet howToMode)
{
	hipError_t err = hipSuccess;
	int nPixel = nx*ny*nz;

	switch(imageMode) {
		case mrcFloatImage: {
			float* inD;
			float* outD;
			float* matD;

			// inD
			DEBUGPRINT3("%d %p -> %p\n", nPixel, in, out);
			err = hipMalloc((void **)&inD,  sizeof(float)*nPixel);
			if(err !=  hipSuccess) {
				fprintf(stderr, "Memory Not Allocated\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMalloc\n");
			}
			// outD
			err = hipMalloc((void **)&outD,  sizeof(float)*nPixel);
			if(err !=  hipSuccess) {
				fprintf(stderr, "Memory Not Allocated\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMalloc\n");
			}
			// matD 
			err = hipMalloc((void **)&matD,  sizeof(float)*4*4);
			if(err !=  hipSuccess) {
				fprintf(stderr, "Memory Not Allocated\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMalloc\n");
			}
			
			// in -> inD
			hipMemcpy(inD, in, sizeof(float)*nPixel, hipMemcpyHostToDevice);
			// inCoord -> inD
			hipMemcpy(matD, mat, sizeof(float)*4*4, hipMemcpyHostToDevice);

			if(err !=  hipSuccess) {
				fprintf(stderr, "hipMemcpy\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMemcpy: HostToDevice\n");
			}
			mrcPixelDataGetbyMatrix3DKernel<<<nPixel/__eosCudaThreadsPerBlock__,__eosCudaThreadsPerBlock__>>>(outD, matD, inD, nx, ny, nz, howToMode);
			
			hipMemcpy(out, outD, sizeof(float)*nPixel, hipMemcpyDeviceToHost);
			if(err !=  hipSuccess) {
				fprintf(stderr, "hipMemcpy\n");
				exit(EXIT_FAILURE);
			} else {
				DEBUGPRINT("Success hipMemcpy: DeviceToHost\n");
			}
			hipFree(inD);
			hipFree(matD);
			hipFree(outD);
			break;
		}
		default: {
			fprintf(stderr, "Not supported mode in mrcPixelDataGetCUDA: %d\n", imageMode);
			exit(EXIT_FAILURE);
			break;
		}
	}

}

