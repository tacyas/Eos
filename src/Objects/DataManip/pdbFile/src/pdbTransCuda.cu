#include "hip/hip_runtime.h"
/*
# pdbTrans.c  1.1
# The latest update : 07/13/96 at 11:39:21
#
#@(#) pdbTrans ver 1.1
#@(#) Created by 
#@(#)
#@(#) Usage : pdbTrans 
#@(#) Attention
#@(#)
*/
//static char __sccs_id[] = "@(#)pdbTrans ver1.1; Date:96/07/13 @(#)";

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/times.h>

#undef DEBUG
extern "C" {
#include "../inc/pdbFile.h"
#include "String.h"
#include "eosCuda.h"
#include "genUtil.h"
#include "Vector.h"
#include "Memory.h"
#include "Matrix3D.h"
};
// pdbTransKernel
// pdbTransXXXKernel: __global__
// pdbTrasnKernelYYY: __device__  
// pdbTrans: __host__

__global__ void pdbTransKernel(float* v, float* mat, int max)
{
	int offset = threadIdx.x + blockIdx.x*blockDim.x;
	float vx, vy, vz, w;
	if(offset<max) {
		vx = v[offset*4+0];
		vy = v[offset*4+1];
		vz = v[offset*4+2];
		w  = v[offset*4+3];

		v[offset*4+0]=mat[0+4*0]*vx + mat[0+4*1]*vy + mat[0+4*2]*vz + mat[0+4*3]*w;
		v[offset*4+1]=mat[1+4*0]*vx + mat[1+4*1]*vy + mat[1+4*2]*vz + mat[1+4*3]*w;
		v[offset*4+2]=mat[2+4*0]*vx + mat[2+4*1]*vy + mat[2+4*2]*vz + mat[2+4*3]*w;
	}
}

void
//pdbTransCuda(pdbFile* pdb, Matrix3D Matrix)
pdbTransCuda(float* hv, int nAtom, Matrix3D Matrix)
{
	//int i;
	//int nAtom;
	//float* hv;
	float* dv;
	float* dmat;
	hipError_t error;
	clock_t t1;
	clock_t t2;

	DEBUGPRINT("pdbTransCuda Start\n");
	DEBUGCOMMAND(t1=clock());
// List to Array 
/*
	nAtom= pdb->nAtom;

	hv    =  (float*)memoryAllocate(sizeof(float)*nAtom*4, "in pdbTransCuda");

	i=0;
	pdbFileTop(pdb);
	DEBUGPRINT2("%p %p\n", pdb->PDB, pdb->top);
	while(NULL!=pdb->PDB) {
		if(pdbFileIsAtom(pdb)) {
			hv[i*4+0] = pdb->PDB->Coord.x; 		
			hv[i*4+1] = pdb->PDB->Coord.y; 		
			hv[i*4+2] = pdb->PDB->Coord.z; 		
			hv[i*4+3] = 1;
		 	DEBUGPRINT3("%f %f %f\n", hv[i*4+0], hv[i*4+1], hv[i*4+2]);
			i++;
		}
		DEBUGPRINT2("%s next %p\n", pdb->PDB->FullRecord, pdb->PDB->next);
	    pdb->PDB = pdb->PDB->next;
	}
	DEBUGPRINT2("%d %d\n", nAtom, i);
	pdb->nAtom = i;
*/

// Array to Cuda 
	error = hipMalloc((void **) &dv, sizeof(float)*nAtom*4);
	if (error != hipSuccess) {
		printf("hipMalloc dv returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	error = hipMalloc((void **) &dmat, sizeof(float)*4*4);
	if (error != hipSuccess) {
		printf("hipMalloc dv returned error code %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(dv, hv, nAtom*4*sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("hipMemcpy %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

	error = hipMemcpy(dmat, (float*)Matrix, 4*4*sizeof(float), hipMemcpyHostToDevice);
	if (error != hipSuccess) {
		printf("hipMemcpy %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}

//
	pdbTransKernel<<<nAtom/__eosCudaThreadsPerBlock__+1, __eosCudaThreadsPerBlock__>>>(dv, dmat, nAtom);
/*
			matrix3DMultiplyVector(&v, Matrix);

			DEBUGPRINT3("%f %f %f\n", Matrix[3][0], Matrix[3][1], Matrix[3][2]);
			DEBUGPRINT2("x %f -> %f\n", pdb->PDB->Coord.x, v.data[0]);
			DEBUGPRINT2("y %f -> %f\n", pdb->PDB->Coord.y, v.data[1]);
			DEBUGPRINT2("z %f -> %f\n", pdb->PDB->Coord.z, v.data[2]);

*/

// Cuda to Array  
	error = hipMemcpy(hv, dv, nAtom*4*sizeof(float), hipMemcpyDeviceToHost);
	if (error != hipSuccess) {
		printf("hipMemcpy %d, line(%d)\n", error, __LINE__);
		exit(EXIT_FAILURE);
	}
	hipFree(dv);
	hipFree(dmat);


// Array to List
/*
	i=0;
	pdbFileTop(pdb);
	DEBUGPRINT2("%p %p\n", pdb->PDB, pdb->top);
	while(NULL!=pdb->PDB) {
		if(pdbFileIsAtom(pdb)) {
		 	DEBUGPRINT3("%f %f %f\n", hv[i*4+0], hv[i*4+1], hv[i*4+2]);
			pdb->PDB->Coord.x = hv[i*4+0]; 		
			pdb->PDB->Coord.y = hv[i*4+1]; 		
			pdb->PDB->Coord.z = hv[i*4+2]; 		
			i++;
	 	}
	    pdb->PDB = pdb->PDB->next;
	}
	memoryFree(hv);
*/
//

	DEBUGCOMMAND(t2=clock());
	DEBUGPRINT1("%ld\n", t2-t1);
	DEBUGPRINT("pdbTransCuda End\n");
}

